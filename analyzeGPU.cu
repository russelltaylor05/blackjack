#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "cpu_poker.h"
#include "poker.h"

void    srand48();
double  drand48();

static void HandleError( hipError_t err, const char * file, int line)
{
  if(err !=hipSuccess){
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))    
 


int main(int argc, char *argv[])
{
  int deck[52];
  int staticHand[HAND_SIZE];
  int blockCnt;
  int analyzeResults[ANALYZE_RESOLUTION];
  int size, sum =0;
  int i, score, rank;
  ARGSP *argsp;
  
  clock_t start, stop;
  
  int *devHand;
  int *devAnalyzeResults;  
  hiprandState *devStates;

  argsp = (ARGSP *)malloc(sizeof(ARGSP));  
  if(getArgs(argsp, argc, argv) < 0) {
    printf("Card arguments broken\n");
    return EXIT_FAILURE;
  };

  srand48((int) time(NULL));  
    
  init_deck_cpu(deck);
  setHandFromArgs(deck, staticHand, argsp);

  score = eval_5hand_cpu(staticHand);
  rank = hand_rank_cpu(score);
  printf("Hand: \t\t");
  print_hand_cpu(staticHand, HAND_SIZE);
  printf("\nScore: \t\t%d\n", score);
  printf("Rank: \t\t%s\n", value_str_cpu[rank]);  

  start = clock();

  // Cuda Memeory Setup
  HANDLE_ERROR(hipMalloc((void **)&devStates, ANALYZE_RESOLUTION * sizeof(hiprandState)));
  
  size = HAND_SIZE * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devHand, HAND_SIZE * sizeof(int)));
  HANDLE_ERROR(hipMemcpy(devHand, staticHand, size, hipMemcpyHostToDevice));

  size = ANALYZE_RESOLUTION * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devAnalyzeResults, size));  

  blockCnt = (ANALYZE_RESOLUTION + THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  analyzeHand<<<blockCnt,THREADS_PER_BLOCK>>>(devHand, devHand, HAND_SIZE, devAnalyzeResults, devStates);
  
  size = ANALYZE_RESOLUTION * sizeof(int);
  HANDLE_ERROR(hipMemcpy(analyzeResults, devAnalyzeResults, size, hipMemcpyDeviceToHost));

  for(i = 0; i < ANALYZE_RESOLUTION; i++) {
    sum +=  analyzeResults[i];
  }
  stop = clock();
  
  printf("Score: \t\t%.2f%%\n", (float)sum / (float)ANALYZE_RESOLUTION * 100.0);
  printf("Time: \t\t%f seconds\n", (double)(stop - start) / CLOCKS_PER_SEC);  

  // Free Cleanup
  hipFree(devAnalyzeResults);
  hipFree(devHand);
  free(argsp);

  return EXIT_SUCCESS;
}

