#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "cpu_poker.h"
#include "poker.h"

static void HandleError( hipError_t err, const char * file, int line)
{
  if(err !=hipSuccess){
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))    
 


int main(int argc, char *argv[])
{
  int deck[52];
  int staticHand[HAND_SIZE];
  int blockCnt;
  int analyzeResults[ANALYZE_RESOLUTION];
  int size, sum =0;
  int i, score, rank;
  ARGSP *argsp;
  
  hipEvent_t start, stop;
  float   elapsedTime;
  
  
  int *devHand;
  int *devAnalyzeResults;  
  hiprandState *devStates;

  argsp = (ARGSP *)malloc(sizeof(ARGSP));  
  if(getArgs(argsp, argc, argv) < 0) {
    printf("Card arguments broken\n");
    return EXIT_FAILURE;
  };

  srand48((int) time(NULL));  
    
  init_deck_cpu(deck);
  setHandFromArgs(deck, staticHand, argsp);

  score = eval_5hand_cpu(staticHand);
  rank = hand_rank_cpu(score);
  printf("Hand: \t\t");
  print_hand_cpu(staticHand, HAND_SIZE);
  printf("\nScore: \t\t%d\n", score);
  printf("Rank: \t\t%s\n", value_str_cpu[rank]);  
  printf("Analyze Res: \t%d\n", ANALYZE_RESOLUTION);  

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  // Cuda Memeory Setup
  HANDLE_ERROR(hipMalloc((void **)&devStates, ANALYZE_RESOLUTION * sizeof(hiprandState)));
  
  size = HAND_SIZE * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devHand, HAND_SIZE * sizeof(int)));
  HANDLE_ERROR(hipMemcpy(devHand, staticHand, size, hipMemcpyHostToDevice));

  size = ANALYZE_RESOLUTION * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devAnalyzeResults, size));  


  HANDLE_ERROR(hipMalloc((void **)&devStates, ANALYZE_RESOLUTION * sizeof(hiprandState)));  

  blockCnt = (ANALYZE_RESOLUTION + THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  analyzeHand<<<blockCnt,THREADS_PER_BLOCK>>>(devHand, devHand, HAND_SIZE, devAnalyzeResults, devStates);

  
  HANDLE_ERROR(hipEventRecord( stop, 0 ));
  HANDLE_ERROR(hipEventSynchronize( stop ));
  HANDLE_ERROR(hipEventElapsedTime( &elapsedTime, start, stop ));
  printf( "Kernel Time:  \t%.1f ms\n", elapsedTime );


  size = ANALYZE_RESOLUTION * sizeof(int);
  HANDLE_ERROR(hipMemcpy(analyzeResults, devAnalyzeResults, size, hipMemcpyDeviceToHost));

  for(i = 0; i < ANALYZE_RESOLUTION; i++) {
    sum +=  analyzeResults[i];
  }
  
  printf("Score: \t\t%.2f%%\n", (float)sum / (float)ANALYZE_RESOLUTION * 100.0);
  //printf("Time: \t\t%f seconds\n", (double)(stop - start) / CLOCKS_PER_SEC);  

  // Free Cleanup
  hipFree(devAnalyzeResults);
  hipFree(devHand);
  free(argsp);

  return EXIT_SUCCESS;
}

