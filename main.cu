#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand.h>
#include "poker.h"

static void HandleError( hipError_t err, const char * file, int line)
{
  if(err !=hipSuccess){
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



int main(int argc, char *argv[])
{

  int *dev_a, *a;
  
  a = (int *) malloc(sizeof(int));
  
  HANDLE_ERROR(hipMalloc(&dev_a, sizeof(int)));

  analyze<<<1,10>>>(1);
  
  HANDLE_ERROR(hipMemcpy(a, dev_a, sizeof(int), hipMemcpyDeviceToHost));

  return 0;
}