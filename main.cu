#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand.h>
#include "lookuptable.h"
#include "poker.h"

static void HandleError( hipError_t err, const char * file, int line)
{
  if(err !=hipSuccess){
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



int main(int argc, char *argv[])
{

  int *dev_a, *a;
  
  a = (int *) malloc(sizeof(int));
  
  HANDLE_ERROR(hipMalloc(&dev_a, sizeof(int)));

  analyze<<<1,10>>>(1);
  
  HANDLE_ERROR(hipMemcpy(a, dev_a, sizeof(int), hipMemcpyDeviceToHost));

  return 0;
}


__global__ void analyze(int i)
{
  int deck[52], hand[HAND_SIZE], staticHand[HAND_SIZE];
  int score, rank;
  int throwAwayCnt;
  int throwAwayCards[HAND_SIZE * 2];
  float results;
  int cardIndex = 0;

  init_deck(deck);

  cardIndex = find_card(Nine, DIAMOND, deck);
  hand[0] = deck[cardIndex];  
  cardIndex = find_card(Ace, HEART, deck);
  hand[1] = deck[cardIndex];
  cardIndex = find_card(Ten, SPADE, deck);
  hand[2] = deck[cardIndex];
  cardIndex = find_card(Queen, HEART, deck);
  hand[3] = deck[cardIndex];
  cardIndex = find_card(King, HEART, deck);
  hand[4] = deck[cardIndex];

  print_hand(hand, HAND_SIZE);
  printf("\n");
}




// perform a binary search on a pre-sorted array
//
__device__ int findit( int key )
{
    int low = 0, high = 4887, mid;

    while ( low <= high )
    {
        mid = (high+low) >> 1;      // divide by two
        if ( key < products[mid] )
            high = mid - 1;
        else if ( key > products[mid] )
            low = mid + 1;
        else
            return( mid );
    }
    printf( "ERROR:  no match found; key = %d\n", key );
    
    return( -1 );
}

//
//   This routine initializes the deck.  A deck of cards is
//   simply an integer array of length 52 (no jokers).  This
//   array is populated with each card, using the following
//   scheme:
//
//   An integer is made up of four bytes.  The high-order
//   bytes are used to hold the rank bit pattern, whereas
//   the low-order bytes hold the suit/rank/prime value
//   of the card.
//
//   +--------+--------+--------+--------+
//   |xxxbbbbb|bbbbbbbb|cdhsrrrr|xxpppppp|
//   +--------+--------+--------+--------+
//
//   p = prime number of rank (deuce=2,trey=3,four=5,five=7,...,ace=41)
//   r = rank of card (deuce=0,trey=1,four=2,five=3,...,ace=12)
//   cdhs = suit of card
//   b = bit turned on depending on rank of card
//
void init_deck( int *deck )
{
    int i, j, n = 0, suit = 0x8000;

    for ( i = 0; i < 4; i++, suit >>= 1 )
        for ( j = 0; j < 13; j++, n++ )
            deck[n] = primes[j] | (j << 8) | suit | (1 << (16+j));
}


//  This routine will search a deck for a specific card
//  (specified by rank/suit), and return the INDEX giving
//  the position of the found card.  If it is not found,
//  then it returns -1
//
__device__ int find_card( int rank, int suit, int *deck )
{
	int i, c;

	for ( i = 0; i < 52; i++ ) {
		c = deck[i];
		if ( (c & suit)  &&  (RANK(c) == rank) )
			return( i );
	}
	return( -1 );
}


//
//  This routine takes a deck and randomly mixes up
//  the order of the cards.
//
__device__ void shuffle_deck( int *deck )
{
    int i, n, temp[52];

    for ( i = 0; i < 52; i++ ) {
      temp[i] = deck[i];
    }

    for ( i = 0; i < 52; i++ ) {
        do {
            n = (int)(51.9999999 * 1); //drand48());
        } while ( temp[n] == 0 );
        deck[i] = temp[n];
        temp[n] = 0;
    }
}


__device__ void print_hand( int *hand, int n )
{
  int i, r;
  char suit;
  char *rank = "23456789TJQKA";

  for ( i = 0; i < n; i++ )  {
    r = (*hand >> 8) & 0xF;
    if ( *hand & 0x8000 )
        suit = 'c';
    else if ( *hand & 0x4000 )
        suit = 'd';
    else if ( *hand & 0x2000 )
        suit = 'h';
    else
        suit = 's';

    printf( "%c%c ", rank[r], suit );
    hand++;
  }
}

__device__ void print_card(int card) 
{
  int r;
  char suit;
  char *rank = "23456789TJQKA";

  r = (card >> 8) & 0xF;
  if ( card & 0x8000 )
      suit = 'c';
  else if ( card & 0x4000 )
      suit = 'd';
  else if ( card & 0x2000 )
      suit = 'h';
  else
      suit = 's';

  printf( "%c%c ", rank[r], suit );
}


__device__ int hand_rank( short val )
{
    if (val > 6185) return(HIGH_CARD);        // 1277 high card
    if (val > 3325) return(ONE_PAIR);         // 2860 one pair
    if (val > 2467) return(TWO_PAIR);         //  858 two pair
    if (val > 1609) return(THREE_OF_A_KIND);  //  858 three-kind
    if (val > 1599) return(STRAIGHT);         //   10 straights
    if (val > 322)  return(FLUSH);            // 1277 flushes
    if (val > 166)  return(FULL_HOUSE);       //  156 full house
    if (val > 10)   return(FOUR_OF_A_KIND);   //  156 four-kind
    return(STRAIGHT_FLUSH);                   //   10 straight-flushes
}


__device__ short eval_5cards( int c1, int c2, int c3, int c4, int c5 )
{
  int q;
  short s;

  q = (c1|c2|c3|c4|c5) >> 16;

  /* check for Flushes and StraightFlushes */
  if ( c1 & c2 & c3 & c4 & c5 & 0xF000 )
    return( flushes[q] );

  /* check for Straights and HighCard hands */
  s = unique5[q];
  if ( s )  return ( s );

  /* let's do it the hard way  */
  q = (c1&0xFF) * (c2&0xFF) * (c3&0xFF) * (c4&0xFF) * (c5&0xFF);
  q = findit( q );

  return( values[q] );
}


__device__ short eval_5hand( int *hand )
{
  int c1, c2, c3, c4, c5;

  c1 = *hand++;
  c2 = *hand++;
  c3 = *hand++;
  c4 = *hand++;
  c5 = *hand;

  return( eval_5cards(c1,c2,c3,c4,c5) );
}


// This is a non-optimized method of determining the
// best five-card hand possible out of seven cards.
// I am working on a faster algorithm.
//
__device__ short eval_7hand( int *hand )
{
    int i, j, q, best = 9999, subhand[5];

	for ( i = 0; i < 21; i++ ) {
		for ( j = 0; j < 5; j++ ) {
			subhand[j] = hand[ perm7[i][j] ];
		}
		q = eval_5hand( subhand );
		if ( q < best )
			best = q;
	}
	return( best );
}
