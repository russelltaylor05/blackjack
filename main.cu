#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "poker.h"

static void HandleError( hipError_t err, const char * file, int line)
{
  if(err !=hipSuccess){
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)



int main(int argc, char *argv[])
{
  int deck[52], staticHand[HAND_SIZE];
  float analyzeResults[ANALYZE_RESOLUTION];
  int size;
  //int throwAwayCards[10];
  
  int *devHand;
  float *devAnalyzeResults;
  
  hiprandState *devStates;
  
  CUDA_CALL(hipMalloc((void **)&devStates, 10 * sizeof(hiprandState)));
  
  /* initialize the deck */
  init_deck(deck);
  
  /* Set Hands */
  setStaticHand(deck, staticHand);  
  //setRandomHand(deck, randomHand, throwAwayCards, 0);   
  
  size = HAND_SIZE * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devHand, HAND_SIZE * sizeof(int)));
  HANDLE_ERROR(hipMemcpy(devHand, staticHand, size, hipMemcpyHostToDevice));

  size = ANALYZE_RESOLUTION * sizeof(float);
  HANDLE_ERROR(hipMalloc(&devAnalyzeResults, size));  

  analyzeHand<<<1,2>>>(devHand, devHand, HAND_SIZE, devAnalyzeResults, devStates);
  
  size = ANALYZE_RESOLUTION * sizeof(float);
  HANDLE_ERROR(hipMemcpy(analyzeResults, devAnalyzeResults, size, hipMemcpyDeviceToHost));


  return 0;
}