#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand.h>
#include "poker.h"

static void HandleError( hipError_t err, const char * file, int line)
{
  if(err !=hipSuccess){
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



int main(int argc, char *argv[])
{
  int deck[52], randomHand[HAND_SIZE], staticHand[HAND_SIZE];
  float analyzeResults[ANALYZE_RESOLUTION];
  int size;
  int throwAwayCards[10];
  
  int *devDeck;
  int *devHand;
  float *devAnalyzeResults;
  
  /* initialize the deck */
  init_deck(deck);
  
  /* Set Hands */
  setStaticHand(deck, staticHand);  
  //setRandomHand(deck, randomHand, throwAwayCards, 0);   


  print_hand(staticHand, HAND_SIZE);
  printf("\n");
  
  size = HAND_SIZE * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devHand, HAND_SIZE * sizeof(int)));
  HANDLE_ERROR(hipMemcpy(devHand, staticHand, size, hipMemcpyHostToDevice));

  size = 52 * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devDeck, 52 * sizeof(int)));
  HANDLE_ERROR(hipMemcpy(devDeck, deck, size, hipMemcpyHostToDevice));

  size = ANALYZE_RESOLUTION * sizeof(float);
  HANDLE_ERROR(hipMalloc(&devAnalyzeResults, size));  

  analyzeHand<<<1,10>>>(devHand, devDeck, devHand, HAND_SIZE, devAnalyzeResults);
  
  size = ANALYZE_RESOLUTION * sizeof(float);
  HANDLE_ERROR(hipMemcpy(analyzeResults, devAnalyzeResults, size, hipMemcpyDeviceToHost));


  return 0;
}