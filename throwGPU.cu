#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "cpu_poker.h"
#include "poker.h"

static void HandleError( hipError_t err, const char * file, int line)
{
  if(err !=hipSuccess){
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))    
 

int main(int argc, char *argv[])
{
  int deck[52];
  int staticHand[HAND_SIZE];
  //int tempHand[HAND_SIZE];
  int throwAway[HAND_SIZE];
  int throwCombosResults[THROWAWAY_RESOLUTION * HAND_SIZE];
  int *throwResults;
  int score, rank, throwCnt, size, i;
  int comboBlockCnt, analyzeBlockCnt;
  int sum = 0;
  ARGSP *argsp;
  
  clock_t start, stop;
  
  int *devHand;
  int *devThrowCards;
  int *devThrowCombosResults; 
  int *devThrowResults; 
  hiprandState *devStates;

  argsp = (ARGSP *)malloc(sizeof(ARGSP));  
  if(getArgs(argsp, argc, argv) < 0) {
    printf("Card arguments broken\n");
    return EXIT_FAILURE;
  }; 

  srand48((int) time(NULL));  
    
  init_deck_cpu(deck);
  setHandFromArgs(deck, staticHand, argsp);
  setThrowFromArgs(deck, throwAway, &throwCnt, argsp);

  score = eval_5hand_cpu(staticHand);
  rank = hand_rank_cpu(score);
  printf("Hand: \t\t");    print_hand_cpu(staticHand, HAND_SIZE);
  printf("\nThrow: \t\t"); print_hand_cpu(throwAway, throwCnt);
  printf("\nScore: \t\t%d\n", score);
  printf("Rank: \t\t%s\n", value_str_cpu[rank]);  
  printf("ThrowAway Res: \t%d\n", THROWAWAY_RESOLUTION);  

  start = clock();

  // Cuda Memeory Setup
  analyzeBlockCnt = (THROWAWAY_RESOLUTION * ANALYZE_RESOLUTION + THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  comboBlockCnt = (THROWAWAY_RESOLUTION + THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  
  throwResults = (int *)malloc(analyzeBlockCnt * sizeof(int)); 
  if (throwResults == NULL) {
    fprintf(stderr, "failed to allocate memory.\n");
    return -1;
  }
   
  
  HANDLE_ERROR(hipMalloc((void **)&devStates, THROWAWAY_RESOLUTION * ANALYZE_RESOLUTION * sizeof(hiprandState)));  

  size = HAND_SIZE * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devHand, HAND_SIZE * sizeof(int)));
  HANDLE_ERROR(hipMemcpy(devHand, staticHand, size, hipMemcpyHostToDevice));  

  size = throwCnt * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devThrowCards, throwCnt * sizeof(int)));
  HANDLE_ERROR(hipMemcpy(devThrowCards, throwAway, size, hipMemcpyHostToDevice));

  size = THROWAWAY_RESOLUTION * HAND_SIZE * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devThrowCombosResults, size));

  size = analyzeBlockCnt * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devThrowResults, size));
  HANDLE_ERROR(hipMemset(devThrowResults, 0, size));


  // Kernel Calls
  curandSetup<<<analyzeBlockCnt,THREADS_PER_BLOCK>>>(devStates);     
  
  printf("K1 blockcnt: \t%d\n",comboBlockCnt);
  printf("K1 threadcnt: \t%d\n\n", comboBlockCnt * THREADS_PER_BLOCK);
  createThrowCombos<<<comboBlockCnt,THREADS_PER_BLOCK>>>(devHand, devThrowCards, throwCnt, devThrowCombosResults, devStates);    

  printf("K2 blockcnt: \t%d\n", analyzeBlockCnt);
  printf("K2 threadcnt: \t%d\n", analyzeBlockCnt * THREADS_PER_BLOCK);
  analyzeThrowCombos<<<analyzeBlockCnt,THREADS_PER_BLOCK>>>(devHand, devThrowCombosResults, devThrowResults, devStates);


  // Return Results 
  size = THROWAWAY_RESOLUTION * HAND_SIZE * sizeof(int);
  //HANDLE_ERROR(hipMemcpy(throwCombosResults, devThrowCombosResults, size, hipMemcpyDeviceToHost));

  size = analyzeBlockCnt * sizeof(int);
  HANDLE_ERROR(hipMemcpy(throwResults, devThrowResults, size, hipMemcpyDeviceToHost));

  for(i = 0; i < analyzeBlockCnt; i++) {
    sum += throwResults[i];
  }


  /*
  printf("CPU Combos: \n");
  for (i = 0; i < THROWAWAY_RESOLUTION * HAND_SIZE; i++) {
    for(j = 0; j < 5; j++) {
      tempHand[j] = throwCombosResults[i];
      i++;      
    }
    i--;
    score = eval_5hand_cpu(tempHand);
    rank = hand_rank_cpu(score);
    print_hand_cpu(tempHand, HAND_SIZE);
    printf("\t%d", score);
    printf("\t%s", value_str_cpu[rank]);
    printf("\n");
  }
  */
  

  stop = clock();
  printf("Sum: \t\t%d\n",sum);
  //printf("Result Size: \t%d\n", THROWAWAY_RESOLUTION);
  printf("throwScore: \t%.2f%%\n", (float)sum / (float)(analyzeBlockCnt) * 100.0);
  printf("Time: \t\t%f seconds\n", (double)(stop - start) / CLOCKS_PER_SEC);  

  /*
  HANDLE_ERROR(hipFree(devStates));
  HANDLE_ERROR(hipFree(devHand));
  HANDLE_ERROR(hipFree(devThrowCards));
  HANDLE_ERROR(hipFree(devThrowCombosResults));
  HANDLE_ERROR(hipFree(devThrowResults));
  free(argsp);
  */

  return EXIT_SUCCESS;
}

