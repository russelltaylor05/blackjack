#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "cpu_poker.h"
#include "poker.h"

static void HandleError( hipError_t err, const char * file, int line)
{
  if(err !=hipSuccess){
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))    
 

int main(int argc, char *argv[])
{
  int deck[52];
  int staticHand[HAND_SIZE];
  //int tempHand[HAND_SIZE];
  int throwAway[HAND_SIZE];
  int throwCombosResults[THROWAWAY_RESOLUTION * HAND_SIZE];
  int *throwResults;
  int score, rank, throwCnt, size, blockCnt, i;
  int sum = 0;
  ARGSP *argsp;
  
  clock_t start, stop;
  
  int *devHand;
  int *devThrowCards;
  int *devThrowCombosResults; 
  int *devThrowResults; 
  hiprandState *devStates;

  argsp = (ARGSP *)malloc(sizeof(ARGSP));  
  if(getArgs(argsp, argc, argv) < 0) {
    printf("Card arguments broken\n");
    return EXIT_FAILURE;
  };
  
  throwResults = (int *)malloc(THROWAWAY_RESOLUTION * ANALYZE_RESOLUTION * sizeof(int)); 
  if (throwResults == NULL) {
    fprintf(stderr, "failed to allocate memory.\n");
    return -1;
  }
  

  srand48((int) time(NULL));  
    
  init_deck_cpu(deck);
  setHandFromArgs(deck, staticHand, argsp);
  setThrowFromArgs(deck, throwAway, &throwCnt, argsp);

  score = eval_5hand_cpu(staticHand);
  rank = hand_rank_cpu(score);
  printf("Hand: \t\t");    print_hand_cpu(staticHand, HAND_SIZE);
  printf("\nThrow: \t\t"); print_hand_cpu(throwAway, throwCnt);
  printf("\nScore: \t\t%d\n", score);
  printf("Rank: \t\t%s\n", value_str_cpu[rank]);  
  printf("ThrowAway Res: \t%d\n", THROWAWAY_RESOLUTION);  

  start = clock();

  // Cuda Memeory Setup
  HANDLE_ERROR(hipMalloc((void **)&devStates, THROWAWAY_RESOLUTION * ANALYZE_RESOLUTION * sizeof(hiprandState)));  

  size = HAND_SIZE * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devHand, HAND_SIZE * sizeof(int)));
  HANDLE_ERROR(hipMemcpy(devHand, staticHand, size, hipMemcpyHostToDevice));  

  size = throwCnt * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devThrowCards, throwCnt * sizeof(int)));
  HANDLE_ERROR(hipMemcpy(devThrowCards, throwAway, size, hipMemcpyHostToDevice));

  size = THROWAWAY_RESOLUTION * HAND_SIZE * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devThrowCombosResults, size));

  size = THROWAWAY_RESOLUTION * ANALYZE_RESOLUTION * sizeof(int);
  HANDLE_ERROR(hipMalloc(&devThrowResults, size));  


  // Kernel Calls
  __global__ void curandSetup(hiprandState *state);
  blockCnt = (THROWAWAY_RESOLUTION * ANALYZE_RESOLUTION + THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  curandSetup<<<blockCnt,THREADS_PER_BLOCK>>>(devStates);     
  
  blockCnt = (THROWAWAY_RESOLUTION + THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  printf("K1 blockcnt: \t%d\n",blockCnt);
  printf("K1 threadcnt: \t%d\n\n", blockCnt * THREADS_PER_BLOCK);
  //createThrowCombos<<<blockCnt,THREADS_PER_BLOCK>>>(devHand, devThrowCards, throwCnt, devThrowCombosResults, devStates);    

  blockCnt = (THROWAWAY_RESOLUTION * ANALYZE_RESOLUTION + THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  printf("K2 blockcnt: \t%d\n", blockCnt);
  printf("K2 threadcnt: \t%d\n", blockCnt * THREADS_PER_BLOCK);
  //analyzeThrowCombos<<<blockCnt,THREADS_PER_BLOCK>>>(devHand, devThrowCombosResults, devThrowResults, devStates);


  // Return Results 
  size = THROWAWAY_RESOLUTION * HAND_SIZE * sizeof(int);
  HANDLE_ERROR(hipMemcpy(throwCombosResults, devThrowCombosResults, size, hipMemcpyDeviceToHost));

  size = THROWAWAY_RESOLUTION * ANALYZE_RESOLUTION * sizeof(int);
  HANDLE_ERROR(hipMemcpy(throwResults, devThrowResults, size, hipMemcpyDeviceToHost));

  for(i = 0; i < THROWAWAY_RESOLUTION * ANALYZE_RESOLUTION; i++) {
    sum += throwResults[i];
    //printf("%d, ",throwResults[i]);
  }


  /*
  printf("CPU Combos: \n");
  for (i = 0; i < THROWAWAY_RESOLUTION * HAND_SIZE; i++) {
    for(j = 0; j < 5; j++) {
      tempHand[j] = throwCombosResults[i];
      i++;      
    }
    i--;
    score = eval_5hand_cpu(tempHand);
    rank = hand_rank_cpu(score);
    print_hand_cpu(tempHand, HAND_SIZE);
    printf("\t%d", score);
    printf("\t%s", value_str_cpu[rank]);
    printf("\n");
  }
  */
  

  stop = clock();
  printf("Sum: \t\t%d\n",sum);
  //printf("Result Size: \t%d\n", THROWAWAY_RESOLUTION);
  printf("throwScore: \t%.2f%%\n", (float)sum / (float)(THROWAWAY_RESOLUTION * ANALYZE_RESOLUTION) * 100.0);
  printf("Time: \t\t%f seconds\n", (double)(stop - start) / CLOCKS_PER_SEC);  

  HANDLE_ERROR(hipFree(devStates));
  HANDLE_ERROR(hipFree(devHand));
  HANDLE_ERROR(hipFree(devThrowCards));
  HANDLE_ERROR(hipFree(devThrowCombosResults));
  HANDLE_ERROR(hipFree(devThrowResults));
  free(argsp);

  return EXIT_SUCCESS;
}

