
#include <hip/hip_runtime.h>
#include <stdio.h>

static void HandleError( hipError_t err,
    const char *file,
    int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__device__ int multem( int a, int b ) {
  return a * b;
}

__global__ void mult( int a, int b, int *c ) {
  *c = multem( a, b );
}

int main( void ) {
  int c;
  int *dev_c;
  HANDLE_ERROR( hipMalloc( (void**)&dev_c, sizeof(int) ) );

  mult<<<1,1>>>( 2, 7, dev_c );

  HANDLE_ERROR( hipMemcpy( &c, dev_c, sizeof(int),
        hipMemcpyDeviceToHost ) );
  printf( "2 * 7 = %d\n", c );
  HANDLE_ERROR( hipFree( dev_c ) );

  return 0;
}

